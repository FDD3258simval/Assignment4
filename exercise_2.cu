
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define ARRAY_SIZE 100000
#define TPB 256

double mysecond(){
  struct timeval tp;
  struct timezone tzp;

  gettimeofday(&tp,&tzp);
  return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void saxpy(int n, float *x, float *y, float a){
	for (int i=0; i < n; i++){
		y[i] = a*x[i]+y[i];
	}
}

__global__ void saxpy_gpu(float *xg, float *yg, float a){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	yg[i] = a*xg[i]+yg[i];
}


int main()
{
	float x[ARRAY_SIZE];
	float y[ARRAY_SIZE];
   float z[ARRAY_SIZE];
	float a = 3.0;

   float *xg = 0;
	float *yg = 0;

   double t_CPU, ts_CPU, t_GPU, ts_GPU;

   for (int i=0; i < ARRAY_SIZE; i++){
		x[i] = 4.0;
      y[i] = 2.0;
	}

// Execution on GPU
	hipMalloc(&xg, ARRAY_SIZE*sizeof(float));
	hipMalloc(&yg, ARRAY_SIZE*sizeof(float));

	hipMemcpy(xg, x, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(yg, y, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);

	printf("Computing SAXPY on the GPU... ");
   ts_GPU = mysecond(); 
   saxpy_gpu<<<(ARRAY_SIZE+TPB-1)/TPB, TPB>>>(xg, yg, a);
   hipDeviceSynchronize();
   t_GPU = mysecond()-ts_GPU;
	printf("Done\n");
   hipMemcpy(z, yg, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(yg);
   int bl = ceil(ARRAY_SIZE/TPB);
// Execution on CPU
	printf("Computing SAXPY on the CPU... ");
	ts_CPU = mysecond();
   saxpy(ARRAY_SIZE, x, y, a);
   t_CPU = mysecond()-ts_CPU;
	printf("Done\n");
	
// Compare CPU and GPU result 
	double errg = 1e-10;
   for (int i=0; i < ARRAY_SIZE; i++){
      double err = abs(y[i]-z[i]);
		if (err > errg) errg = err;
	}
   printf("GPU Execution time: %12.9f\n", t_GPU); 
   printf("CPU Execution time: %12.9f\n", t_CPU);
	printf("Maximum error is: %16.12f\n", errg);
   printf("Elements of z: %16.12f, %16.12f\n", z[0], z[ARRAY_SIZE/2]);
   printf("Elements of y: %16.12f, %16.12f\n", y[0], y[ARRAY_SIZE/2]);
	printf("number of blocks: %d\n", bl);
	return 0;
}
